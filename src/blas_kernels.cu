#include "hip/hip_runtime.h"
extern "C" {
#include "blas.h"
#include "hip/hip_runtime.h"
#include "utils.h"
}

__global__ void axpy_kernel(int N, float ALPHA, float *X, int OFFX, int INCX,  float *Y, int OFFY, int INCY)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) Y[OFFY+i*INCY] += ALPHA*X[OFFX+i*INCX];
}

__global__ void pow_kernel(int N, float ALPHA, float *X, int INCX, float *Y, int INCY)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) Y[i*INCY] = pow(X[i*INCX], ALPHA);
}

__global__ void const_kernel(int N, float ALPHA, float *X, int INCX)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) X[i*INCX] = ALPHA;
}

__global__ void scal_kernel(int N, float ALPHA, float *X, int INCX)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) X[i*INCX] *= ALPHA;
}

__global__ void mask_kernel(int n,  float *x, float mask_num, float *mask)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < n && mask[i] == mask_num) x[i] = mask_num;
}

__global__ void copy_kernel(int N,  float *X, int OFFX, int INCX, float *Y, int OFFY, int INCY)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) Y[i*INCY + OFFY] = X[i*INCX + OFFX];
}

__global__ void mul_kernel(int N, float *X, int INCX, float *Y, int INCY)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) Y[i*INCY] *= X[i*INCX];
}

extern "C" void axpy_ongpu(int N, float ALPHA, float * X, int INCX, float * Y, int INCY)
{
    axpy_ongpu_offset(N, ALPHA, X, 0, INCX, Y, 0, INCY);
}

extern "C" void pow_ongpu(int N, float ALPHA, float * X, int INCX, float * Y, int INCY)
{
    pow_kernel<<<cuda_gridsize(N), BLOCK>>>(N, ALPHA, X, INCX, Y, INCY);
    check_error(hipPeekAtLastError());
}

extern "C" void axpy_ongpu_offset(int N, float ALPHA, float * X, int OFFX, int INCX, float * Y, int OFFY, int INCY)
{
    axpy_kernel<<<cuda_gridsize(N), BLOCK>>>(N, ALPHA, X, OFFX, INCX, Y, OFFY, INCY);
    check_error(hipPeekAtLastError());
}

extern "C" void copy_ongpu(int N, float * X, int INCX, float * Y, int INCY)
{
    copy_ongpu_offset(N, X, 0, INCX, Y, 0, INCY);
}

extern "C" void mul_ongpu(int N, float * X, int INCX, float * Y, int INCY)
{
    mul_kernel<<<cuda_gridsize(N), BLOCK>>>(N, X, INCX, Y, INCY);
    check_error(hipPeekAtLastError());
}

extern "C" void copy_ongpu_offset(int N, float * X, int OFFX, int INCX, float * Y, int OFFY, int INCY)
{
    copy_kernel<<<cuda_gridsize(N), BLOCK>>>(N, X, OFFX, INCX, Y, OFFY, INCY);
    check_error(hipPeekAtLastError());
}

extern "C" void mask_ongpu(int N, float * X, float mask_num, float * mask)
{
    mask_kernel<<<cuda_gridsize(N), BLOCK>>>(N, X, mask_num, mask);
    check_error(hipPeekAtLastError());
}

extern "C" void const_ongpu(int N, float ALPHA, float * X, int INCX)
{
    const_kernel<<<cuda_gridsize(N), BLOCK>>>(N, ALPHA, X, INCX);
    check_error(hipPeekAtLastError());
}

extern "C" void scal_ongpu(int N, float ALPHA, float * X, int INCX)
{
    scal_kernel<<<cuda_gridsize(N), BLOCK>>>(N, ALPHA, X, INCX);
    check_error(hipPeekAtLastError());
}
